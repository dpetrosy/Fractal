#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include "fractal.hpp"

__global__ void computeFractalCUDA(sf::Color* pixels, ComplexNumber c,
    double zoom, double offsetX, double offsetY, RGBColor color, FractalType type)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WINDOW_SIZE && y < WINDOW_SIZE)
    {
        c.re = x / zoom + offsetX;
        c.im = y / zoom + offsetY;

        size_t iter = 0;
		switch (type)
		{
		case FractalType::Mandelbrot:
			iter = calcMandelbrotCUDA(c);
			break;
		case FractalType::Julia:
			iter = calcJuliaCUDA(c);
			break;
		case FractalType::BurningShip:
			iter = calcBurningShipCUDA(c);
			break;
		case FractalType::Tricorn:
			iter = calcTricornCUDA(c);
			break;
		case FractalType::Mandelbox:
			iter = calcMandelboxCUDA(c);
			break;
		case FractalType::Multibrot:
			iter = calcMultibrotCUDA(c);
			break;
		case FractalType::Mandelbar:
			iter = calcMandelbarCUDA(c);
			break;
		case FractalType::PerBurningShip:
			iter = calcPerBurningShipCUDA(c);
			break;
		case FractalType::CelticMandelbrot:
			iter = calcCelticMandelbrotCUDA(c);
			break;
		default:
			break;
		}

        pixels[y * WINDOW_SIZE + x].r = color.r * iter;
        pixels[y * WINDOW_SIZE + x].g = color.g * iter;
        pixels[y * WINDOW_SIZE + x].b = color.b * iter;
        pixels[y * WINDOW_SIZE + x].a = 255;
    }
}

void colorizePixelsByGPU(sf::Image& image, Fractal& fractal)
{
    sf::Color* imagePtr;
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&imagePtr, WINDOW_SIZE *
        WINDOW_SIZE * sizeof(sf::Color));

    if (cudaStatus != hipSuccess)
        throw std::runtime_error("-- hipMalloc error: " +
            std::string(hipGetErrorString(cudaStatus)));

    // 256 threads per block
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((WINDOW_SIZE + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (WINDOW_SIZE + threadsPerBlock.y - 1) / threadsPerBlock.y);

    double zoom = fractal.getZoom();
    double offsetX = fractal.getOffsetX();
    double offsetY = fractal.getOffsetY();
    RGBColor& color = fractal.getColor();
    FractalType type = fractal.getType();
    ComplexNumber& c = fractal.getC();

    if (fractal.isNeedToHandleMouseMoved())
    {
        c.reJulia = fractal.getMouseX() / zoom + offsetX;
        c.imJulia = fractal.getMouseY() / zoom + offsetY;
    }

    computeFractalCUDA<<<blocksPerGrid, threadsPerBlock>>>(imagePtr, c, zoom,
        offsetX, offsetY, color, type);
    hipDeviceSynchronize();

    cudaStatus = hipMemcpy((void *)image.getPixelsPtr(), imagePtr, WINDOW_SIZE *
        WINDOW_SIZE * sizeof(sf::Color), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess)
        throw std::runtime_error("-- hipMemcpy error: " +
            std::string(hipGetErrorString(cudaStatus)));
    hipFree(imagePtr);
}
