#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

__device__ size_t calcMandelbrotCUDA(ComplexNumber& c)
{
	ComplexNumber z;
	double reTemp;
    size_t i = 0;

	for (; (z.re * z.re + z.im * z.im < 4.0) && i < MAX_ITER; ++i)
	{
		reTemp = z.re * z.re - z.im * z.im + c.re;
		z.im = 2.0 * z.re * z.im + c.im;
		z.re = reTemp;
	}
	return (i);
}

__device__ size_t calcJuliaCUDA(ComplexNumber& c)
{
	ComplexNumber z = c;
	double reTemp;
    size_t i = 0;

	for (; (z.re * z.re + z.im * z.im < 4.0) && i < MAX_ITER; ++i)
	{
		reTemp = z.re * z.re - z.im * z.im + c.reJulia;
		z.im = 2.0 * z.re * z.im + c.imJulia;
		z.re = reTemp;
	}
	return (i);
}

__device__ size_t calcBurningShipCUDA(ComplexNumber& c)
{
	ComplexNumber z;
	double reTemp;
    size_t i = 0;

	for (; (z.re * z.re + z.im * z.im < 4.0) && i < MAX_ITER; ++i)
	{
		reTemp = z.re * z.re - z.im * z.im + c.re;
		z.im = std::fabs(2.0 * z.re * z.im) + c.im;
		z.re = std::fabs(reTemp);
	}
	return (i);
}

__device__ size_t calcTricornCUDA(ComplexNumber& c)
{
	ComplexNumber z;
	double reTemp;
    size_t i = 0;

	for (; (z.re * z.re + z.im * z.im < 4.0) && i < MAX_ITER; ++i)
	{
		reTemp = z.re * z.re - z.im * z.im + c.re;
		z.im = -2.0 * z.re * z.im + c.im;
		z.re = reTemp;
	}
	return (i);
}

__device__ size_t calcMultibrotCUDA(ComplexNumber& c)
{
	ComplexNumber z;
	double reSqr;
	double imSqr;
    size_t i = 0;

	for (; (z.re * z.re + z.im * z.im < 4.0) && i < MAX_ITER; ++i)
	{
		reSqr = z.re * z.re;
		imSqr = z.im * z.im;
		z.re = z.re * (reSqr*reSqr - 10.0*reSqr*imSqr + 5.0*imSqr*imSqr) + c.re;
		z.im = z.im * (5.0*reSqr*reSqr - 10.0*reSqr*imSqr + imSqr*imSqr) + c.im;
	}
	return (i);
}

__device__ size_t calcMandelbarCUDA(ComplexNumber& c)
{
	ComplexNumber z;
	double reSqr;
	double imSqr;
    size_t i = 0;

	for (; (z.re * z.re + z.im * z.im < 4.0) && i < MAX_ITER; ++i)
	{
		reSqr = z.re * z.re;
		imSqr = z.im * z.im;
		z.re = -z.re * (reSqr*reSqr - 10.0*reSqr*imSqr + 5.0*imSqr*imSqr) + c.re;
		z.im = z.im * (5.0*reSqr*reSqr - 10.0*reSqr*imSqr + imSqr*imSqr) + c.im;
	}
	return (i);
}

__device__ size_t calcPerBurningShipCUDA(ComplexNumber& c)
{
	ComplexNumber z;
	double reTemp;
    size_t i = 0;

	for (; (z.re * z.re + z.im * z.im < 4.0) && i < MAX_ITER; ++i)
	{
		reTemp = z.re * z.re - z.im * z.im + c.re;
		z.im = 2.0 * z.re * std::fabs(z.im) + c.im;
		z.re = reTemp;
	}
	return (i);
}

__device__ size_t calcCelticMandelbrotCUDA(ComplexNumber& c)
{
	ComplexNumber z;
	double reTemp;
    size_t i = 0;

	for (; (z.re * z.re + z.im * z.im < 4.0) && i < MAX_ITER; ++i)
	{
		reTemp = std::fabs(z.re * z.re - z.im * z.im) + c.re;
		z.im = 2.0 * z.re * z.im + c.im;
		z.re = reTemp;
	}
	return (i);
}

__device__ double boxFoldCUDA(double z)
{
	if (z > 1.0)
		z = 2.0 - z;
	else if (z < -1.0)
		z = -2.0 - z;
	return (z);
}

__device__ double ballFoldCUDA(double r, double mag)
{
	if (mag < r)
		mag = mag / (r * r);
	else if (mag < 1.0)
		mag = 1.0 / (mag * mag);
	return (mag);
}

__device__ size_t calcMandelboxCUDA(ComplexNumber& c)
{
	ComplexNumber z = c;
	double mag = 0.0;
    size_t i = 0;

	for (; sqrtf(mag) < 2.0 && i < MAX_ITER; ++i)
	{
		z.re = FIXED_RADIUS * boxFoldCUDA(z.re);
		z.im = FIXED_RADIUS * boxFoldCUDA(z.im);
		mag = sqrtf(z.re * z.re + z.im * z.im);
		z.re = z.re * SCALE * ballFoldCUDA(MINIMUM_RADIUS, mag) + c.re;
		z.im = z.im * SCALE * ballFoldCUDA(MINIMUM_RADIUS, mag) + c.im;
	}
	return (i);
}
